#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>

#include "rotate_rect_ops.h"

// TODO make it in a common file
#define CUDA_1D_KERNEL_LOOP(i, n)                            \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
       i += blockDim.x * gridDim.x)


template <typename T>
__device__ T bilinear_interpolate(const T* bottom_data,
    const int height, const int width,
    T y, T x,
    const int index /* index for debug only*/) {

  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    //empty
    return 0;
  }

  if (y <= 0) y = 0;
  if (x <= 0) x = 0;

  int y_low = (int) y;
  int x_low = (int) x;
  int y_high;
  int x_high;

  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = (T) y_low;
  } else {
    y_high = y_low + 1;
  }

  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = (T) x_low;
  } else {
    x_high = x_low + 1;
  }

  T ly = y - y_low;
  T lx = x - x_low;
  T hy = 1. - ly, hx = 1. - lx;
  // do bilinear interpolation
  T v1 = bottom_data[y_low * width + x_low];
  T v2 = bottom_data[y_low * width + x_high];
  T v3 = bottom_data[y_high * width + x_low];
  T v4 = bottom_data[y_high * width + x_high];
  T w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

  return val;
}



template <typename T>
__device__ void bilinear_interpolate_gradient(
    const int height, const int width,
    T y, T x,
    T & w1, T & w2, T & w3, T & w4,
    int & x_low, int & x_high, int & y_low, int & y_high,
    const int index /* index for debug only*/) {

  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    //empty
    w1 = w2 = w3 = w4 = 0.;
    x_low = x_high = y_low = y_high = -1;
    return;
  }

  if (y <= 0) y = 0;
  if (x <= 0) x = 0;

  y_low = (int) y;
  x_low = (int) x;

  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = (T) y_low;
  } else {
    y_high = y_low + 1;
  }

  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = (T) x_low;
  } else {
    x_high = x_low + 1;
  }

  T ly = y - y_low;
  T lx = x - x_low;
  T hy = 1. - ly, hx = 1. - lx;

  // reference in forward
  // T v1 = bottom_data[y_low * width + x_low];
  // T v2 = bottom_data[y_low * width + x_high];
  // T v3 = bottom_data[y_high * width + x_low];
  // T v4 = bottom_data[y_high * width + x_high];
  // T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

  w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  return;
}


template <typename T>
__global__ void RRoIAlignForward(const int nthreads, const T* bottom_data,
    const T spatial_scale, const int channels,
    const int height, const int width,
    const int pooled_height, const int pooled_width,
    const int sampling_ratio,
    const T* bottom_rois, T* top_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;


    const T* offset_bottom_rois = bottom_rois + n * 6;  // batch_ind, xc, yc, w, h, angle
    int roi_batch_ind = offset_bottom_rois[0];

    // Force malformed ROIs to be 1x1
    T roi_width = max(offset_bottom_rois[3] * spatial_scale, (T)1.);
    T roi_height = max(offset_bottom_rois[4] * spatial_scale, (T)1.);

    const T* offset_bottom_data = bottom_data + (roi_batch_ind * channels + c) * height * width;

    // We use roi_bin_grid to sample the grid and mimic integral
    int roi_bin_grid_h = (sampling_ratio > 0) ? sampling_ratio : ceil(roi_height / pooled_height); // e.g., = 2
    int roi_bin_grid_w = (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);

    const T mw = 1.0 / roi_bin_grid_w;
    const T mh = 1.0 / roi_bin_grid_h;

    // compute pool points
    T P[8];
    compute_roi_pool_pts(offset_bottom_rois, P, spatial_scale, pooled_height, pooled_width, ph, pw);

    // compute line params
    T line_params[4];
    for (int i = 0; i < 2; ++i)
    {
        line_params[i * 2] = P[((i + 1) * 2) % 8] - P[i * 2];
        line_params[i * 2 + 1] = P[((i + 1) * 2) % 8 + 1] - P[i * 2 + 1];
    }

    // We do average (integral) pooling inside a bin
    const T count = roi_bin_grid_h * roi_bin_grid_w; // e.g. = 4

    T output_val = 0.;
    for (int iy = 0; iy < roi_bin_grid_h; iy ++) // e.g., iy = 0, 1
    {
      for (int ix = 0; ix < roi_bin_grid_w; ix ++)
      {
        const T x = P[0] + static_cast<T>(iy + 0.5) * line_params[0] * mh + static_cast<T>(ix + 0.5) * line_params[2] * mw;
        const T y = P[1] + static_cast<T>(iy + 0.5) * line_params[1] * mh + static_cast<T>(ix + 0.5) * line_params[3] * mw;

        T val = bilinear_interpolate(offset_bottom_data, height, width, y, x, index);
        output_val += val;
//        printf("%.2f\n", val);
      }
    }

    output_val /= count;

    top_data[index] = output_val;
  }
}


template <typename T>
__global__ void RRoIAlignBackwardFeature(const int nthreads, const T* top_diff,
    const int num_rois, const T spatial_scale,
    const int channels, const int height, const int width,
    const int pooled_height, const int pooled_width,
    const int sampling_ratio,
    T* bottom_diff,
    const T* bottom_rois) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const T* offset_bottom_rois = bottom_rois + n * 6;
    int roi_batch_ind = offset_bottom_rois[0];

    // Force malformed ROIs to be 1x1
    T roi_width = max(offset_bottom_rois[3] * spatial_scale, (T)1.);
    T roi_height = max(offset_bottom_rois[4] * spatial_scale, (T)1.);

    // We use roi_bin_grid to sample the grid and mimic integral
    int roi_bin_grid_h = (sampling_ratio > 0) ? sampling_ratio : ceil(roi_height / pooled_height); // e.g., = 2
    int roi_bin_grid_w = (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);

    const T mw = 1.0 / roi_bin_grid_w;
    const T mh = 1.0 / roi_bin_grid_h;

    T* offset_bottom_diff = bottom_diff + (roi_batch_ind * channels + c) * height * width;

    int top_offset    = (n * channels + c) * pooled_height * pooled_width;
    const T* offset_top_diff = top_diff + top_offset;
    const T top_diff_this_bin = offset_top_diff[ph * pooled_width + pw];

    // compute pool points
    T P[8];
    compute_roi_pool_pts(offset_bottom_rois, P, spatial_scale, pooled_height, pooled_width, ph, pw);

    // compute line params
    T line_params[4];
    for (int i = 0; i < 2; ++i)
    {
        line_params[i * 2] = P[((i + 1) * 2) % 8] - P[i * 2];
        line_params[i * 2 + 1] = P[((i + 1) * 2) % 8 + 1] - P[i * 2 + 1];
    }

    // We do average (integral) pooling inside a bin
    const T count = roi_bin_grid_h * roi_bin_grid_w; // e.g. = 4

    for (int iy = 0; iy < roi_bin_grid_h; iy ++) // e.g., iy = 0, 1
    {
      for (int ix = 0; ix < roi_bin_grid_w; ix ++)
      {
        const T x = P[0] + static_cast<T>(iy + 0.5) * line_params[0] * mh + static_cast<T>(ix + 0.5) * line_params[2] * mw;
        const T y = P[1] + static_cast<T>(iy + 0.5) * line_params[1] * mh + static_cast<T>(ix + 0.5) * line_params[3] * mw;

        T w1, w2, w3, w4;
        int x_low, x_high, y_low, y_high;

        bilinear_interpolate_gradient(height, width, y, x,
            w1, w2, w3, w4,
            x_low, x_high, y_low, y_high,
            index);

        T g1 = top_diff_this_bin * w1 / count;
        T g2 = top_diff_this_bin * w2 / count;
        T g3 = top_diff_this_bin * w3 / count;
        T g4 = top_diff_this_bin * w4 / count;

        if (x_low >= 0 && x_high >= 0 && y_low >= 0 && y_high >= 0)
        {
          atomicAdd(offset_bottom_diff + y_low * width + x_low, static_cast<T>(g1));
          atomicAdd(offset_bottom_diff + y_low * width + x_high, static_cast<T>(g2));
          atomicAdd(offset_bottom_diff + y_high * width + x_low, static_cast<T>(g3));
          atomicAdd(offset_bottom_diff + y_high * width + x_high, static_cast<T>(g4));
        } // if
      } // ix
    } // iy
  } // CUDA_1D_KERNEL_LOOP
} // RoIAlignBackward




at::Tensor RROIAlign_forward_cuda(const at::Tensor& input,
                                const at::Tensor& rois,
                                const float spatial_scale,
                                const int pooled_height,
                                const int pooled_width,
                                const int sampling_ratio)
{
  AT_ASSERTM(input.type().is_cuda(), "input must be a CUDA tensor");
  AT_ASSERTM(rois.type().is_cuda(), "rois must be a CUDA tensor");

  auto num_rois = rois.size(0);
  auto channels = input.size(1);
  auto height = input.size(2);
  auto width = input.size(3);

  auto output = at::empty({num_rois, channels, pooled_height, pooled_width}, input.options());
  auto output_size = num_rois * pooled_height * pooled_width * channels;
//  auto argmax = at::zeros({num_rois, channels, pooled_height, pooled_width}, input.options().dtype(at::kInt));

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(std::min(THCCeilDiv((long)output_size, 512L), 4096L));
  dim3 block(512);

  if (output.numel() == 0) {
    THCudaCheck(hipGetLastError());
    return output;
  }

  AT_DISPATCH_FLOATING_TYPES(input.type(), "RROIAlign_forward", [&] {
    RRoIAlignForward<scalar_t><<<grid, block, 0, stream>>>(
         output_size,
         input.contiguous().data<scalar_t>(),
         spatial_scale,
         channels,
         height,
         width,
         pooled_height,
         pooled_width,
         sampling_ratio,
         rois.contiguous().data<scalar_t>(),
         output.data<scalar_t>()
     );
  });
  THCudaCheck(hipGetLastError());
  return output;
}

at::Tensor RROIAlign_backward_cuda(const at::Tensor& grad,
                      const at::Tensor& rois,
                      const float spatial_scale,
                      const int pooled_height,
                      const int pooled_width,
                      const int batch_size,
                      const int channels,
                      const int height,
                      const int width,
                      const int sampling_ratio)
{
  AT_ASSERTM(grad.type().is_cuda(), "grad must be a CUDA tensor");
  AT_ASSERTM(rois.type().is_cuda(), "rois must be a CUDA tensor");

  auto num_rois = rois.size(0);
  auto grad_input = at::zeros({batch_size, channels, height, width}, grad.options());

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(std::min(THCCeilDiv((long)grad.numel(), 512L), 4096L));
  dim3 block(512);

  // handle possibly empty gradients
  if (grad.numel() == 0) {
    THCudaCheck(hipGetLastError());
    return grad_input;
  }

  AT_DISPATCH_FLOATING_TYPES(grad.type(), "RROIAlign_backward", [&] {
    RRoIAlignBackwardFeature<scalar_t><<<grid, block, 0, stream>>>(
        grad.numel(),
        grad.contiguous().data<scalar_t>(),
        num_rois,
        spatial_scale,
        channels,
        height,
        width,
        pooled_height,
        pooled_width,
        sampling_ratio,
        grad_input.data<scalar_t>(),
        rois.contiguous().data<scalar_t>());
  });
  THCudaCheck(hipGetLastError());
  return grad_input;
}
